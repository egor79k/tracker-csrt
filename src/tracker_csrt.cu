#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "tracker_csrt.hpp"


typedef float2 Complex;


// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}


TrackerCSRT::TrackerCSRT(const cv::Mat& frame, const cv::Rect& bbox) :
    filter(frame, bbox),
    channels(1),
    channelWeights(1)
{
    size_t realSize = bbox.width * bbox.height * sizeof(float);
    size_t complexSize = bbox.width * bbox.height * sizeof(hipfftComplex);

    hipMalloc(&dSrc, realSize);
    hipMalloc(&dKernel, realSize);
    hipMalloc(&dSrcFFT, complexSize);
    hipMalloc(&dKernelFFT, complexSize);
}


TrackerCSRT::~TrackerCSRT() {
    hipFree(dSrc);
    hipFree(dKernel);
    hipFree(dSrcFFT);
    hipFree(dKernelFFT);
}


void TrackerCSRT::getChannelFilter(const int channel_id, cv::Mat1f& filter) {
    // Calculate the closed-form solution for channel filter using formula (4)
}


// void TrackerCSRT::convolve(const cv::Mat1f& src, const cv::Mat1f& kernel, cv::Mat1f& dst) {
//     cv::filter2D(src, dst, -1, kernel, cv::Point(-1, -1), 0, cv::BORDER_WRAP);
// }


void TrackerCSRT::convolveCUDA(const cv::Mat1f& src, const cv::Mat1f& kernel, cv::Mat1f& dst) {
    // The optimal size for DFT transform
    size_t width = src.cols;
    size_t height = src.rows;

    cv::Size dftSize;
    dftSize.width = width;
    dftSize.height = height;

    cv::Mat1f srcTemp = cv::Mat1f::zeros(dftSize);
    cv::Mat1f kernelTemp = cv::Mat1f::zeros(dftSize);

    // Copy src and kernel to the top-left corners of srcTemp and kernelTemp
    cv::Mat srcRoi(srcTemp, cv::Rect(0, 0, src.cols, src.rows));
    cv::Mat kernelRoi(kernelTemp, cv::Rect(0, 0, kernel.cols, kernel.rows));

    src.copyTo(srcRoi);
    kernel.copyTo(kernelRoi);

    // Copy src and kernel to device
    hipMemcpy2D(dSrc, width * sizeof(float), srcTemp.data, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);
    hipMemcpy2D(dKernel, width * sizeof(float), kernelTemp.data, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);

    // Apply FFT to srcTemp and kernel
    hipfftHandle plan1;
    hipfftPlan2d(&plan1, height, width, HIPFFT_R2C);
    hipfftExecR2C(plan1, dSrc, dSrcFFT);
    hipfftExecR2C(plan1, dKernel, dKernelFFT);
    hipfftDestroy(plan1);

    // Multiply elementwise srcTemp and kernel in Fourier domain
    ComplexPointwiseMulAndScale<<<32, 256>>>(dSrcFFT, dKernelFFT, width * height, 1.0f / (width * height));

    // Apply IFFT to srcTemp
    hipfftHandle plan2;
    hipfftPlan2d(&plan2, height, width, HIPFFT_C2R);
    hipfftExecC2R(plan2, dSrcFFT, dSrc);
    hipfftDestroy(plan2);

    // Copy result to host
    dst = cv::Mat1f::zeros(height, width);
    hipMemcpy2D(dst.data, width * sizeof(float), dSrc, width * sizeof(float), width * sizeof(float), height, hipMemcpyDeviceToHost);

    // cv::normalize(dst, dst, 0, 1, cv::NORM_MINMAX);
    // cv::imshow("Copy", dst);
    // cv::waitKey(0);
}


void TrackerCSRT::convolveOpenCV(const cv::Mat1f& src, const cv::Mat1f& kernel, cv::Mat1f& dst) {
    // The optimal size for DFT transform
    cv::Size dftSize;

    dftSize.width = cv::getOptimalDFTSize(src.cols + kernel.cols);
    dftSize.height = cv::getOptimalDFTSize(src.rows + kernel.rows);

    cv::Mat1f srcTemp = cv::Mat1f::zeros(dftSize);
    cv::Mat1f kernelTemp = cv::Mat1f::zeros(dftSize);

    // Copy src and kernel to the top-left corners of srcTemp and kernelTemp
    cv::Mat srcRoi(srcTemp, cv::Rect(0, 0, src.cols, src.rows));
    cv::Mat kernelRoi(kernelTemp, cv::Rect(0, 0, kernel.cols, kernel.rows));

    src.copyTo(srcRoi);
    kernel.copyTo(kernelRoi);
    
    // Transform src and kernel into frequency domain
    cv::dft(srcTemp, srcTemp, 0, src.rows);
    cv::dft(kernelTemp, kernelTemp, 0, kernel.rows);
    
    // Multiply matrices elementwise in frequency domain
    cv::mulSpectrums(srcTemp, kernelTemp, srcTemp, 0);
    
    // Transform result from the frequency domain
    cv::idft(srcTemp, srcTemp);
    
    // Copy result to dst
    cv::Rect convRoi(
        kernel.cols,
        kernel.rows,
        cv::abs(src.cols - kernel.cols),
        cv::abs(src.rows - kernel.rows));

    srcTemp(convRoi).copyTo(dst);
}


bool TrackerCSRT::update(const cv::Mat& frame, cv::Rect& bbox) {
    // Temporary use just grayscale channel with 1.0 weight
    cv::Mat1f(frame, bbox).convertTo(channels[0], CV_32F);
    channelWeights[0] = 1.0f;

    // cv::Rect wrappedRoi(bbox.width / 2, bbox.height / 2, bbox.width * 2, bbox.height * 2);
    cv::Mat1f resultingConvolution = cv::Mat::zeros(bbox.height, bbox.width, CV_32F);
    cv::Mat1f channelConvolution;

    for (int channel_id = 0; channel_id < channels.size(); ++channel_id) {
        getChannelFilter(channel_id, filter);

        // Replicate image to caclulate cyclic convolution
        // cv::Mat1f wrappedChannel(cv::repeat(channels[channel_id], 3, 3), wrappedRoi);

        // convolveCUDA(wrappedChannel, filter, channelConvolution);
        convolveCUDA(channels[channel_id], filter, channelConvolution);

        // Add weighted channel convolution to resulting response
        resultingConvolution = resultingConvolution + channelConvolution * channelWeights[channel_id];
    }

    // Find the location of maximum in convolution response
    double minVal;
    double maxVal;
    cv::Point minLoc;
    cv::Point maxLoc;
    cv::minMaxLoc(resultingConvolution, &minVal, &maxVal, &minLoc, &maxLoc);

    // Move the bounding box according to maximum
    bbox += (minLoc - cv::Point(bbox.width / 2, bbox.height / 2));
    // cv::Mat1f(frame, bbox).convertTo(filter, CV_32F);

    return true;
}
